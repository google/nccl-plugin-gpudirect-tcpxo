#include "hip/hip_runtime.h"
/*
* Copyright 2025 Google LLC
*
* Use of this source code is governed by a BSD-style
* license that can be found in the LICENSE.md file or at
* https://developers.google.com/open-source/licenses/bsd
 */

#include "buffer_mgmt_daemon/cuda_logging.h"

#include "absl/log/log.h"
#include "absl/strings/str_format.h"

namespace tcpdirect {

bool CUCallSuccess(hipError_t err) {
  if (err != hipSuccess) {
    const char* name = nullptr;
    const char* reason = nullptr;
    if (hipDrvGetErrorName(err, &name)) {
      LOG(FATAL) << "Error: error getting error name from CU error " << err;
      return false;
    }
    if (hipDrvGetErrorString(err, &reason)) {
      LOG(FATAL) << "Error: error getting error string from CU error " << err;
      return false;
    }
    LOG(ERROR) << absl::StrFormat("cuda error detected! name: %s; string: %s",
                                  name, reason);
    return false;
  }
  return true;
}

bool CUDACallSuccess(hipError_t err) {
  if (err != hipSuccess) {
    const char* name = hipGetErrorName(err);
    const char* reason = hipGetErrorString(err);
    if (name == nullptr || reason == nullptr) {
      LOG(FATAL) << "Failed to get error name and reason from CUDA error "
                 << err;
      return false;
    }
    LOG(ERROR) << absl::StrFormat("cuda error detected! name: %s; string: %s",
                                  name, reason);
    return false;
  }
  return true;
}

}  // namespace tcpdirect
